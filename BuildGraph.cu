#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdbool.h>



// map gnome seq to graph
// read k-mer from file


/*
    mapping
    A -> 0;
    T -> 1;
    G -> 2;
    C -> 3;

*/

int getIndex(char* kmer){
    int ans = 0;
    for(int i=0;i<strlen(kmer);i++){

    }

    return ans;
}


// each k-mer starts at 1,8,16,... mapping  array for edges

//takes comnd line arg K (k-mer)
int main(int argc,char* argv){

    int K;
    if(argc < 2){
        printf("Argument K missing !!\n");
        exit(0);
    }

    K = atoi(argv[1]);
    int size = (int)pow(4,K);       //all possible k-mers 

    //array that stores if the node is valid or not i.e present or not
    bool* h_node = (bool*)malloc(sizeof(bool)*size);

    File* fptr = fopen("data.txt","r");

    char buffer[100];
    while(fscanf(fptr,"%s\n",buffer)!=NULL){
        if(strlen(buffer)==K){
            int idx = getIndex(buffer);
            h_node[idx] = true;
        }else{

        }
    }

    // output of gpu kernel stored in it
    //array of int* each 8 consecutive block belongs to a node
    int* h_edge = (int*)malloc(sizeof(int)*size*8);
    
    //****for gpu*******

    int* d_edge;
    hipMalloc(&d_edge,size*8*sizeof(int));
    bool* d_node;
    hipMalloc(&d_node,size*sizeof(int));

    //copy node array to gpu
    hipMemcpy(d_node,h_node,size*sizeof(int),hipMemcpyHostToDevice);

    //set all val to -1 
    hipMemset(d_edge,-1,size*8*sizeof(int));

    //invoke kernel

    //end kernel

    hipMemcpy(h_edge,d_edge,size*8*sizeof(int),hipMemcpyDeviceToHost);

    //node contains valid node and its adjacency info in 
    // edge array .


    //TODO
    // Traverse the graph and build the assembly
    



    
    
    



}