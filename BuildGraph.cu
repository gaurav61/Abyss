#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <string.h>
#include <math.h>
#include <stdbool.h>

// map gnome seq to graph
// read k-mer from file


/*
    mapping
    A -> 0;
    T -> 1;
    G -> 2;
    C -> 3;

*/

typedef struct Pair_{
    unsigned long a;
    unsigned long b;
    unsigned long c;
    unsigned long d;
} Pair;



Pair genPair(char* kmer){
	Pair ans;
	ans.a =0;ans.b=0;
	ans.c=0;ans.d=0;
	
	int count = 0;
	int index = strlen(kmer) -1;
	unsigned long shiftop = 1; 
	for( ;count<32 && index>=0;index--){
		switch(kmer[index]){
			case 'A': ans.d+=shiftop*0;    
					break;
			case 'T': ans.d+=shiftop*1;
					break;
			case 'G': ans.d+=shiftop*2;
					break;
			case 'C': ans.d+=shiftop*3;
					break;
		}
		shiftop=shiftop<<2;     //multiply by 4
		//  printf("%lu \n",shiftop);
		count++;
	}

	shiftop = 1;
	count = 0;
	for( ;count<32 && index>=0;index--){
		switch(kmer[index]){
			case 'A': ans.c+=shiftop*0;    
					break;
			case 'T': ans.c+=shiftop*1;
					break;
			case 'G': ans.c+=shiftop*2;
					break;
			case 'C': ans.c+=shiftop*3;
					break;
		}
		shiftop=shiftop<<2;       //multiply by 4
		count++;
	}

	shiftop = 1;
	count = 0;
	for( ;count<32 && index>=0;index--){
		switch(kmer[index]){
			case 'A': ans.b+=shiftop*0;    
					break;
			case 'T': ans.b+=shiftop*1;
					break;
			case 'G': ans.b+=shiftop*2;
					break;
			case 'C': ans.b+=shiftop*3;
					break;
		}
		shiftop=shiftop<<2;       //multiply by 4
		count++;
	}

	shiftop = 1;
	count = 0;
	for( ;count<32 && index>=0;index--){
		switch(kmer[index]){
			case 'A': ans.a+=shiftop*0;    
					break;
			case 'T': ans.a+=shiftop*1;
					break;
			case 'G': ans.a+=shiftop*2;
                    break;
			case 'C': ans.a+=shiftop*3;
					break;
		}
		shiftop=shiftop<<2;      //multiply by 4
		count++;
	}

	return ans;
}
    




__global__ void buildGraph(unsigned long* Node, unsigned int* Edge, unsigned int N,int SA,int SB,int SC,int SD){
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < N){
        unsigned int k = 4*idx;
        unsigned long a = Node[k]; 
        unsigned long b = Node[k+1]
        unsigned long c = Node[k+2];
        unsigned long d = Node[k+3];

        unsigned int e = 8*idx;

        unsigned long end = 3;
        unsigned long start = end<<62;

        //at begining
        for(unsigned long i=0;i<4;i++){
            //add last base of C to first of D
            //atleast size of 1 base 2 bits
            unsigned long a1,b1,c1,d1;
            a1=b1=c1=d1=0;

            if(SD>1 && SC>1)
                d1 = (d>>2) | ( (c & end)<<(SD-2));
            if(SC>1 && SB>1)
                c1= (c>>2) | ( (b & end)<<(SC-2));
            if(SB>1 && SA>1)
                b1 = (b>>2) | ( (a & end)<<(SB-2))
            
            //Add here diff bases
            if(SA>1)
                a1 = (a>>2) | (i<<(SA-2));
            else if(SB>1)
                b1 = (b>>2) | (i<<(SB-2));
            else if(SC>1)
                c1 = (c>>2) | (i<<(SC-2));
            else if(SD>1)
                d1 = (d>>2) | (i<<(SD-2));

            //search for a1,b1,c1,d1

        }

        //at end
        for(unsigned long i=0;i<4;i++){
            unsigned long a1,b1,c1,d1;
            a1=b1=c1=d1=0;
            unsigned long lim = 1;
            
            if(SA>1)
                a1 = (a<<2) | ( (b & start)>>62 );
            if(SB>1)
                b1 = (b<<2) | ( (c & start)>>62 );
            if(SC>1)
                c1 = (c<<2) | ( (d & start)>>62 );
            if(SD>1)
                d1 = (d<<2)| i ;
            
            //removing additional bits if not full limit
            if(SA>1 && SA<64)
                a1 = a1 & ((lim<<SA)-1);
            if(SA>1 && SA<64)
                b1 = b1 & ((lim<<SB)-1);
            if(SA>1 && SA<64)
                c1 = c1 & ((lim<<SC)-1);
            if(SA>1 && SA<64)
                d1 = d1 & ((lim<<SD)-1);

            //search now
        }
    } 
}
 

// each k-mer starts at 1,8,16,... mapping  array for edges

//takes comnd line arg K (k-mer)
int main(int argc,char* argv){

    int K,N;
    if(argc < 3){
        printf("Argument K missing !!\n");
        exit(0);
    }


    K = atoi(argv[1]); 
    unsigned int N = 100;       //taking it as given
    /********************
    *   Data structure
    *   Node representing a sequence => 4 64bit num
    *   adjacency                    => 8 32bit num
    *
    *********************/
    

    File* fptr = fopen("data.txt","r");

    // Array containing Nodes each 4 digit represents one seq
    unsigned long* Node = (unsigned long*)malloc(sizeof(unsigned long)*N*4);

    // Array contain edges for each node as 8 per Node
    unsigned int*  Edge = (unsigned int*) malloc(sizeof(unsigned int)*N*8);

 
    char buffer[100];
    unsigned int Nindex = 0;
    while(fscanf(fptr,"%s\n",buffer)!=NULL){
        if(strlen(buffer)==K){
            Pair p = genPair(buffer);
            Node[Nindex] = p.a;
            Node[Nindex+1]=p.b;
            Node[Nindex+2]=p.c;
            Node[Nindex+3]=p.d;
            Nindex+=4;
        }else{
            // handel out of order strings
        }

    }
 
    //****for gpu*******

    unsigned long* DNode;
    unsigned long DNsize = sizeof(unsigned long)*N*4;
    
    unsigned int* DEdge;
    unsigned long DEsize = sizeof(unsigned int)*N*8;
    
    hipMalloc(&DNode,DNsize);
    hipMalloc(&DEdge,DEsize);

    //copy node array to gpu
    hipMemcpy(DNode,Node,DNsize,hipMemcpyHostToDevice);

    //set all val to -1 
    hipMemset(DEdge,-1,DEsize);

    //invoke kernel

    int numThreads = 1024;
    int numBlocks = (N+numThreads-1)/numThreads;



    //end kernel

    hipMemcpy(h_edge,d_edge,size*8*sizeof(int),hipMemcpyDeviceToHost);

    //node contains valid node and its adjacency info in 
    // edge array .


    //TODO
    // Traverse the graph and build the assembly
    


}
