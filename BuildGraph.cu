#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdbool.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <string>
#include <unordered_map>
#include <set>
#include <stack>

using namespace std;
// map gnome seq to graph
// read k-mer from file

//GPU KERNEL here
__global__ void buildGraph(unsigned long* Node,unsigned int* Edge,unsigned int N,int SA,int SB,int SC,int SD,unsigned int* stt,unsigned int* enn){
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < N){
        unsigned int k = 4*idx;
        unsigned long a = Node[k]; 
        unsigned long b = Node[k+1];
        unsigned long c = Node[k+2];
        unsigned long d = Node[k+3];

        unsigned long end = 3;
        unsigned long start = end<<62;

        //at begining
        for(unsigned long i=0;i<4;i++){
            //add last base of C to first of D
            //atleast size of 1 base 2 bits
            unsigned long a1,b1,c1,d1;
            a1=b1=c1=d1=0;

            if(SD>1 && SC>1)
                d1 = (d>>2) | ( (c & end)<<(SD-2));
            if(SC>1 && SB>1)
                c1= (c>>2) | ( (b & end)<<(SC-2));
            if(SB>1 && SA>1)
                b1 = (b>>2) | ( (a & end)<<(SB-2));
            
            //Add here diff bases
            if(SA>1)
                a1 = (a>>2) | (i<<(SA-2));
            else if(SB>1)
                b1 = (b>>2) | (i<<(SB-2));
            else if(SC>1)
                c1 = (c>>2) | (i<<(SC-2));
            else if(SD>1)
                d1 = (d>>2) | (i<<(SD-2));

            int m,l,r;
            l=0;
            r=(int)N-1;
		    while ( l <= r) 
		    { 
		        m = l + (r-l)/2; 
		  	
		        // Check if x is present at mid 
		        if (Node[4*m] == a1  && Node[4*m+1] == b1  && Node[4*m+2] == c1  && Node[4*m+3] == d1 ){
		        	Edge[8*idx+i]=m; 
		        	break;
		        }
		        else if(Node[4*m] >a1) r = m-1;
		        else if(Node[4*m] <a1) l = m +1;
		        else{
		        	if(Node[4*m+1] >b1) r=m-1;
		        	else if(Node[4*m+1] <b1) l = m +1;
		        	else{
		        		if(Node[4*m+2] >c1) r=m-1;
		        		else if(Node[4*m+2] <c1) l = m +1;
		        		else{
		        			if(Node[4*m+3] >d1) r=m-1;
		        			else l = m +1;
							//all possible case is already taken care.
		        		}
		        	}

		        }
		        
		    } 
        }
       // if(idx==8)printf("%lu\n",d);
        //at end
        for(unsigned long i=0;i<4;i++){
            unsigned long a1,b1,c1,d1;
            a1=b1=c1=d1=0;
            unsigned long lim = 1;
            
            if(SA>1)
                a1 = (a<<2) | ( (b & start)>>62 );
            if(SB>1)
                b1 = (b<<2) | ( (c & start)>>62 );
            if(SC>1)
                c1 = (c<<2) | ( (d & start)>>62 );
            if(SD>1)
                d1 = (d<<2)| i ;
            
            //removing additional bits if not full limit
            if(SB>1 && SA<64)
                a1 = a1 & ((lim<<SA)-1);
            if(SC>1 && SB<64)
                b1 = b1 & ((lim<<SB)-1);
            if(SD>1 && SC<64)
                c1 = c1 & ((lim<<SC)-1);
            if(SD<64)
                d1 = d1 & ((lim<<SD)-1);

            //search now
            
            int m,l,r;
            l=0;
            r=(int)N-1;
           // if(idx==8) printf("idx:%u\t = a1:%lu\tb1:%lu\tc1:%lu\td1:%lu\n",idx+1 ,a1,b1,c1,d1);
		    while ( l <= r) 
		    { 
		        m = l + (r-l)/2; 
		       // if(idx==8)printf(" %d ",m);
		  
		        // Check if x is present at mid 
		        if (Node[4*m] == a1  && Node[4*m+1] == b1  && Node[4*m+2] == c1  && Node[4*m+3] == d1 ){
		        	Edge[8*idx+4+i]=m; 
		        	
		        
		        //	if(idx==8) printf("found\n");
		        	break;
		        }
		        else if(Node[4*m] >a1) r = m-1;
		        else if(Node[4*m] <a1) l = m +1;
		        else{
		        	if(Node[4*m+1] >b1) r=m-1;
		        	else if(Node[4*m+1] <b1) l = m +1;
		        	else{
		        		if(Node[4*m+2] >c1) r=m-1;
		        		else if(Node[4*m+2] <c1) l = m +1;
		        		else{
		        			if(Node[4*m+3] >d1) r=m-1;
		        			else l = m +1;
							//all possible case is already taken care.
		        		}
		        	}
		        }
		  		

		    }
		    
        }

        if(Edge[idx*8]==UINT_MAX && Edge[idx*8+1]==UINT_MAX && Edge[idx*8+2]==UINT_MAX && Edge[idx*8+3]==UINT_MAX) *stt = idx;
        if(Edge[idx*8+4]==UINT_MAX && Edge[idx*8+5]==UINT_MAX && Edge[idx*8+6]==UINT_MAX && Edge[idx*8+7]==UINT_MAX) *enn = idx;

    } 
}



/*
    mapping
    A -> 0;
    C -> 1;
    G -> 2;
    T -> 3;

*/

// we are representing kmer as as a pair of 4 unsigned long
// at max there can be 128 base pairs in kmer 
typedef struct Pair_{      
    unsigned long a;
    unsigned long b;
    unsigned long c;
    unsigned long d;
} Pair;

class Vertex {
public:
    string contig;
    vector<Vertex*> Edges;

    Vertex(){
        contig="";
    }

    void addContig(string st1){
        contig+=st1[st1.size()-1];
    }

    void addEdge(Vertex* v){
        Edges.push_back(v);
    }
};



//global vars
unordered_map<unsigned int,Vertex*>Graph;
set<unsigned int> Removed;
int SA,SB,SC,SD;

void add_all_edges(unsigned int* Edge,Vertex* v,int i,int N);
void rec_merge(unsigned int* Edge,unsigned long* Node,int N,Vertex* v,int id,int KS);
string getDNA(unsigned long* Node,int id,int N);


Pair genPair(char* kmer){
	Pair ans;
	ans.a =0;ans.b=0;
	ans.c=0;ans.d=0;
	
	unsigned long e;
	int count = 0,i=0;
	int index = strlen(kmer) -1;
	//printf("%d\n",index);
	unsigned long shiftop = 1; 

	//converting rightmost part of kmer into long and storing in d
	for(i=0;i<4 && index>=0 ;i++){
		shiftop =1;
		count=0;
		e=0;
		for(;count<32 && index>=0;index--){
			switch(kmer[index]){
				case 'A': e+=shiftop*0;    
						break;
				case 'C': e+=shiftop*1;
						break;
				case 'G': e+=shiftop*2;
						break;
				case 'T': e+=shiftop*3;
						break;
			}
			shiftop=shiftop<<2;     //multiply by 4
			count++;
		}

		switch(i){
			case 0: ans.d = e;break;
			case 1: ans.c = e;break;
			case 2: ans.b = e;break;
			case 3: ans.a = e;break;
		}
	}

	return ans;
}
     

// each k-mer starts at 1,8,16,... mapping  array for edges

//takes comnd line arg K (k-mer)
int main(int argc,char* argv){

    int K,N;
    if(argc < 3){
        printf("Argument K missing !!\n");
        exit(0);
    }


    K = atoi(argv[1]); 
    unsigned int N = 100;       //taking it as given
    
    int A=0,B=0,C=0,D=0,l;
    int l=2*K;

    for(int i=0;i<4 && l>0 ;i++){
    	int m=0;
    	if(l<=64) {m=l;l=0;}
    	else{ 
    		m=64;
    		l=l-64;
    	}
    	switch(i){
    		case 0: D=m;break;
    		case 1: C=m;break;
    		case 2: B=m;break;
    		case 3: A=m;break;
    	}	
    	
    }

    printf("%d %d %d %d\n",A,B,C,D );
	SA=A;SB=B;SC=C;SD=D;



    File* fptr = fopen("ReadsData.txt","r");

    // Array containing Nodes each 4 digit represents one seq
    unsigned long* Node = (unsigned long*)malloc(sizeof(unsigned long)*N*4);

    // Array contain edges for each node as 8 per Node
    unsigned int*  Edge = (unsigned int*) malloc(sizeof(unsigned int)*N*8);

 
    char buffer[100];
    unsigned int Nindex = 0;
    while(fscanf(fptr,"%s\n",buffer)!=NULL){
        if(strlen(buffer)==K){
            Pair p = genPair(buffer);
            Node[Nindex] = p.a;
            Node[Nindex+1]=p.b;
            Node[Nindex+2]=p.c;
            Node[Nindex+3]=p.d;
            Nindex+=4;
        }else{
            // handel out of order strings
        }

    }

    unsigned int* start = (unsigned int*)malloc(sizeof(unsigned int));
	unsigned int* end = (unsigned int*)malloc(sizeof(unsigned int));

 
    //****for gpu*******
    unsigned long* DNode;
    unsigned long DNsize = sizeof(unsigned long)*N*4;
    
    unsigned int* DEdge;
    unsigned long DEsize = sizeof(unsigned int)*N*8;
    
    unsigned int* Dstart;
    unsigned int* Dend;

    hipMalloc(&DNode,DNsize);
    hipMalloc(&DEdge,DEsize);
    hipMalloc(&Dstart,sizeof(unsigned int));
    hipMalloc(&Dend,sizeof(unsigned int));

    //copy node array to gpu
    hipMemcpy(DNode,Node,DNsize,hipMemcpyHostToDevice);
    hipMemcpy(Dstart,start,sizeof(unsigned int),hipMemcpyHostToDevice);
    hipMemcpy(Dend,end,sizeof(unsigned int),hipMemcpyHostToDevice);


    //set all val to -1 
    hipMemset(DEdge,UINT_MAX,DEsize);

    //invoke kernel

    int numThreads = 1024;
    int numBlocks = (N+numThreads-1)/numThreads;


    buildGraph<<<numBlocks,numThreads>>>(DNode,DEdge,N,SA,SB,SC,SD,Dstart,Dend);

    //end kernel
    hipMemcpy(DEdge,Edge,DEsize,hipMemcpyDeviceToHost);

    //node contains valid node and its adjacency info in 
    // edge array .


    //TODO
    // Traverse the graph and build the assembly
    
	for(unsigned int i=0;i<N;i++){
		
		set<unsigned int>::iterator sit;
		sit=Removed.find(i);
		if(sit!=Removed.end())
			continue;

		Vertex* v;
		unordered_map<unsigned int,Vertex*>::iterator it;
		it = Graph.find(i);
		if(it==Graph.end()){
			v = new Vertex;
			Graph.insert(pair<unsigned int,Vertex*>(i,v));
		}
		else{
			v = it->second;
		}

		//only single outgoing edge
		
		rec_merge(Edge,Node,N,v,i,K);
		
	}

	for(auto sit=Removed.begin();sit!=Removed.end();sit++){
		unordered_map<unsigned int,Vertex*>::iterator it;
		it=Graph.find(*sit);
		if(it!=Graph.end()){
			Graph.erase(it);
		}
	}


		//print graph
	cout<<"Graph\n";
	for(auto it=Graph.begin();it!=Graph.end();it++){
		cout<<it->second->contig<<" :";
		for(int j=0;j<it->second->Edges.size();j++){
			cout<<it->second->Edges[j]->contig<<" , ";
		}
		cout<<"\n";
	}	



	unordered_map<unsigned int,Vertex*>::iterator it2;
	it2=Graph.find(*start);
	if(it2==Graph.end()){
		cout<<"Error\n";
		exit(0);
	}
	Vertex* st = it2->second;
	Vertex* en = NULL;
	for(auto it=Graph.begin();it!=Graph.end();it++){
		if(it->second->Edges.size()==0 && it->second->contig.size()>0)
			en=it->second;
	}	

	if(en==NULL){
		cout<<"Error\n";
		exit(0);
	}
	 

	if(Graph.size()==1){
		for(auto it=Graph.begin();it!=Graph.end();it++){
			cout<<it->second->contig<<"\n";
			return 0;
		}	
	}

	//find the path
	//make dummy edge from end to start
	en->addEdge(st);

	//if more than one vertex
	cout<<"start tarversing\n";
	vector<Vertex*>path;
	stack<Vertex*> curr_path;
	Vertex* curr_v= st;
	curr_path.push(st);

	while(!curr_path.empty()){
		if(curr_v->Edges.size()>0){
			curr_path.push(curr_v);
			Vertex* next_v = curr_v->Edges[curr_v->Edges.size()-1];
			curr_v->Edges.pop_back();

			curr_v=next_v;
		}
		else{
			path.push_back(curr_v);
			curr_v =  curr_path.top();
			curr_path.pop();
		}
	}

	cout<<"path\n";
	for(int i=path.size()-1;i>=0;i--){
		cout<<path[i]->contig<<"\n";
	}

	return 0;



}

void rec_merge(unsigned int* Edge,unsigned long* Node,int N,Vertex* v,int id,int KS){
	
	set<unsigned int>::iterator sit;
	sit=Removed.find(id);
	if(sit!=Removed.end()){
		unsigned int parent;
		for(int j=0;j<4;j++){
			if(Edge[8*id+j]!=UINT_MAX){
				parent=Edge[8*id+j];
				break;
			}
		}
		unordered_map<unsigned int,Vertex*>::iterator itr;
		itr=Graph.find(parent);
		if(itr!=Graph.end()){
			//add all nodes to vertex
			for(int k=0;k<(itr->second)->Edges.size();k++){
				v->addEdge(itr->second->Edges[k]);
			}

			v->contig+=itr->second->contig.substr(KS);

			Graph.erase(itr);
			
			return;
		}

	}
	
	int out=0;
	unsigned int oid;
	for(int j=0;j<4;j++){
		if(Edge[8*id+4+j]!=UINT_MAX && out==0){
			oid=Edge[8*id+4+j];
			out++;
		}
	}


	bool flag_merge = false;

	if(out==1){
		int in=0;
		for(int j=0;j<4;j++){
			if(Edge[8*oid+j]!=UINT_MAX)
				in++;
		}

		if(in==1){
			// cout<<v->contig<<"-";
			//recurse
			if(v->contig.size()==0)
				v->contig = getDNA(Node,id,N);
			else
				v->addContig(getDNA(Node,id,N));
			
			// cout<<v->contig<<"\n";
			rec_merge(Edge,Node,N,v,oid,KS);
			Removed.insert(oid);
			flag_merge=true;
		}

	}

	if(flag_merge==false){	
		// cout<<v->contig<<"-";

		if(v->contig.size()==0)
			v->contig = getDNA(Node,id,N);
		else
			v->addContig(getDNA(Node,id,N));
		
	   // cout<<v->contig<<"\n";

		add_all_edges(Edge,v,id,N);	
	}	

}

void add_all_edges(unsigned int* Edge,Vertex* v,int i,int N){
	if(i>=N)
		return;
	
	for(int j=0;j<4;j++){
		if(Edge[8*i+4+j]!=UINT_MAX){
			Vertex* temp;
			unordered_map<unsigned int,Vertex*>::iterator itr;
			itr = Graph.find(Edge[8*i+4+j]);
			if(itr==Graph.end()){
				temp = new Vertex;
				Graph.insert(pair<unsigned int,Vertex*>(Edge[8*i+4+j],temp));
				
			}
			else{
				temp = itr->second;
				
			}	
			v->addEdge(temp);
		}
	}
}

string getDNA(unsigned long* Node,int id,int N){
	string dna="";
	unsigned long end = 3;
	unsigned long a,b,c,d;
	a=Node[4*id];
	b=Node[4*id+1];
	c=Node[4*id+2];
	d=Node[4*id+3];	
	for(int i=0;i< SD;i+=2){
		unsigned long temp = (d>>i) & end;
		char ch='$';
		switch(temp){
			case 0:ch='A';break;
			case 1:ch='C';break;
			case 2:ch='G';break;
			case 3:ch='T';break;
		}
		dna=ch+dna;
	}

	for(int i=0;i< SC;i+=2){
		unsigned long temp = (c>>i) & end;
		char ch='$';
		switch(temp){
			case 0:ch='A';break;
			case 1:ch='C';break;
			case 2:ch='G';break;
			case 3:ch='T';break;
		}
		dna=ch+dna;
	}

	for(int i=0;i< SB;i+=2){
		unsigned long temp = (b>>i) & end;
		char ch='$';
		switch(temp){
			case 0:ch='A';break;
			case 1:ch='C';break;
			case 2:ch='G';break;
			case 3:ch='T';break;
		}
		dna=ch+dna;
	}

	for(int i=0;i< SA;i+=2){
		unsigned long temp = (a>>i) & end;
		char ch='$';
		switch(temp){
			case 0:ch='A';break;
			case 1:ch='C';break;
			case 2:ch='G';break;
			case 3:ch='T';break;
		}
		dna=ch+dna;
	}

	return dna;
}
