#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdbool.h>



// map gnome seq to graph
// read k-mer from file


/*
    mapping
    A -> 0;
    T -> 1;
    G -> 2;
    C -> 3;

*/

// each k-mer starts at 1,8,16,... mapping  array for edges

//node array if k-mer is valid or not i.e represent a node of graph if 1

int main(int argc,char* argv){

    int K;
    if(argc < 2){
        printf("Argument K missing !!\n");
        exit(0);
    }

    K = atoi(argv[1]);
    int size = (int)pow(4,K);

    //array of node*
    int* d_edge = (int*)malloc(sizeof(int)*size*8);

    bool* d_node = (bool*)malloc(sizeof(bool)*size);

    File* fptr = fopen("data.txt","r");

    char buffer[100];
    while(fscanf(fptr,"%s\n",buffer)!=NULL){
        if(strlen(buffer)==K){
            int idx = getIndex(buffer);
            d_node[idx] = true;
        }else{

        }
    }

    // input as scanned in arrays
    



}